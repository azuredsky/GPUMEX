#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <mex.h> 
#include <stdio.h>


void mexFunction()
{
   
  hipInit(0);



  hipDevice_t dev;

  int nGPUs;

  hipGetDeviceCount(&nGPUs);

  mexPrintf("Device Info: %d GPUs found in system.\n", nGPUs);



  hipCtx_t ctx;

  hipDeviceGet(&dev,0);       // use 1st CUDA device

  hipCtxCreate(&ctx, 0, dev); // create context for it



  hipError_t memres;

  unsigned int free, total;

  memres = hipMemGetInfo(&free, &total);

  mexPrintf("After all allocation(%d):     free(MB) %d     total(MB) %d \n", memres, free/1024/1024, total/1024/1024);



  cuCtxDetach(ctx);
}
